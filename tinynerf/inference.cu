#include <tiny-cuda-nn/misc_kernels.h>

#include <tiny-cuda-nn/config.h>

#include <chrono>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <random>
#include <stdexcept>
#include <string>
#include <thread>
#include <vector>

#include "data.h"
#include "utils.h"
#include "nerf.h"
#include <xtensor/xio.hpp>

using namespace tcnn;
using std::string;
using std::vector;
using precision_t = network_precision_t;

int main(int argc, char *argv[])
{
  // 0. Load precomputed weights into network
  // 1. Load a known pose
  // 2. Run inference with the pose
  // 3. Perform volume rendering to output RGB image
  // 4. Write image to file

  const uint32_t image_width = 100;
  const uint32_t image_height = 100;
  const uint8_t n_frequencies = 6;
  const uint8_t n_samples = 64;
  const float near = 2.;
  const float far = 6.;
  // TinyNeRF only uses (xyz) -- no direction
  const uint32_t n_input_dims = 3;
  const uint32_t n_output_dims = 4; // RGB + density

  // Number of (x, y, z) coordinates to compute
  uint32_t n_coords = image_width * image_height * n_samples;

  try
  {
    json config = {
        {"loss", {{"otype", "L2"}}},
        {"optimizer", {
                          {"otype", "Adam"},
                          {"learning_rate", 1e-5},
                          {"beta1", 0.9f},
                          {"beta2", 0.99f},
                      }},
        {"encoding", {
                         {"otype", "Frequency"},
                         {"n_frequencies", n_frequencies},
                     }},
        {"network", {
                        {"otype", "FullyFusedMLP"},
                        {"n_neurons", 64},
                        {"n_hidden_layers", 8},
                        {"activation", "ReLU"},
                        {"output_activation", "None"},
                    }},
    };

    json encoding_opts = config.value("encoding", json::object());
    json loss_opts = config.value("loss", json::object());
    json optimizer_opts = config.value("optimizer", json::object());
    json network_opts = config.value("network", json::object());

    std::shared_ptr<Loss<precision_t>> loss{create_loss<precision_t>(loss_opts)};
    std::shared_ptr<Optimizer<precision_t>> optimizer{create_optimizer<precision_t>(optimizer_opts)};
    std::shared_ptr<NetworkWithInputEncoding<precision_t>> network = std::make_shared<NetworkWithInputEncoding<precision_t>>(n_input_dims, n_output_dims, encoding_opts, network_opts);

    auto trainer = std::make_shared<Trainer<float, precision_t, precision_t>>(network, optimizer, loss);

    // 0. Load precomputed network weights
    vector<float> weights = load_weights("../nerfdata/tinylego/weights.txt");
    std::cout << "weights: " << weights.size() << std::endl;

    for (auto &p : network->layer_sizes())
    {
      std::cout << p.second << ", " << p.first << std::endl;
    }
    trainer->set_params_full_precision(weights.data(), weights.size());

    // 1. Load a known pose

    json transform_data = read_json("../nerfdata/tinylego/transforms.json");
    string dataset_path = "../nerfdata/tinylego";
    // In TinyNeRF dataset, camer_angle_x is the focal length (no need to calculate)
    float focal_length = transform_data["camera_angle_x"];

    auto [image_paths, c2ws] = get_image_c2w(transform_data, dataset_path);

    auto pose = c2ws[0];
    auto [ray_origins, ray_directions] = get_ray_bundle(image_width, image_height, focal_length, pose);
    auto [query_pts, depth_values] = compute_query_points_from_rays(ray_origins, ray_directions, near, far, n_samples);
    auto pts_flat = flatten_query_pts(query_pts);
    std::vector<float> host_pts_vec(pts_flat.begin(), pts_flat.end());
    GPUMemory<float> pts_vec(host_pts_vec.size());
    pts_vec.copy_from_host(host_pts_vec);

    std::cout << "Total number of (x, y, z) points for inference: " << host_pts_vec.size() / n_input_dims << std::endl;

    hipStream_t inference_stream;
    CUDA_CHECK_THROW(hipStreamCreate(&inference_stream));
    GPUMatrix<float> inference_batch(pts_vec.data(), n_input_dims, n_coords);
    GPUMatrix<float> prediction(n_output_dims, n_coords);

    // 2. Run inference
    network->inference(inference_stream, inference_batch, prediction);
    // Need to move prediction matrix from GPU to CPU
    std::vector<float> host_output(n_coords * n_output_dims);
    CUDA_CHECK_THROW(hipMemcpy(host_output.data(), prediction.data(), host_output.size() * sizeof(float), hipMemcpyDeviceToHost));
    std::cout << "output number of elements: " << host_output.size() << std::endl;

    // 3. Volume rendering
    auto radiance_field = xt::adapt(host_output, {100, 100, 64, 4});
    std::cout << xt::adapt(radiance_field.shape()) << std::endl;
    auto rgb = render_rays(radiance_field, ray_origins, depth_values);
    std::cout << rgb << std::endl;
  }
  catch (std::exception &e)
  {
    std::cout << "Uncaught exception: " << e.what() << std::endl;
  }

  return EXIT_SUCCESS;
}