#include <tiny-cuda-nn/misc_kernels.h>
#include <tiny-cuda-nn/networks/fully_fused_mlp.h>

#include <tiny-cuda-nn/config.h>

#include <chrono>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <random>
#include <stdexcept>
#include <string>
#include <thread>
#include <vector>

#include "data.h"
#include "utils.h"
#include "nerf.h"
#include <xtensor/xio.hpp>
#include <xtensor/xmanipulation.hpp>
#include <xtensor/xoperation.hpp>
#include <algorithm>

using namespace tcnn;
using std::string;
using std::vector;
using precision_t = network_precision_t;

std::ostream &operator<<(std::ostream &o, tcnn::MatrixLayout c)
{
  std::cout << static_cast<int>(c);
  return o;
}

template <typename T>
void render_output(const T *output, int n_coords, int image_height, int image_width, int n_samples, int n_output_dims, xt::xtensor<float, 1UL, xt::layout_type::row_major> &depth_values)
{
  std::vector<T> host_data(n_coords * n_output_dims);
  CUDA_CHECK_THROW(hipMemcpy(host_data.data(), output, host_data.size() * sizeof(T), hipMemcpyDeviceToHost));

  std::vector<float> float_host_data(host_data.size());
  for (size_t i = 0; i < host_data.size(); ++i)
  {
    float_host_data[i] = (float)host_data[i];
  }

  std::vector<std::size_t> rf_shape = {(size_t)image_height, (size_t)image_width, (size_t)n_samples, (size_t)n_output_dims};
  xt::xtensor<float, 4> radiance_field = xt::adapt(float_host_data, rf_shape);
  auto rgb = render_rays(radiance_field, depth_values);

  // auto rgb_8bit = xt::cast<char>(rgb * 255.999);
  // const std::vector<unsigned char> rgb_vec(rgb_8bit.begin(), rgb_8bit.end());
  // encodePNG("output.png", rgb_vec.data(), image_width, image_height);
}

int main(int argc, char *argv[])
{
  // 0. Load precomputed weights into network
  // 1. Load a known pose
  // 2. Run inference with the pose
  // 3. Perform volume rendering to output RGB image
  // 4. Write image to file

  const uint32_t image_width = 100;
  const uint32_t image_height = 100;
  const uint8_t n_frequencies = 6;
  const uint8_t n_samples = 64;
  const float near = 2.;
  const float far = 6.;
  // TinyNeRF only uses (xyz) -- no direction
  const uint32_t n_input_dims = 39;
  const uint32_t n_output_dims = 4; // RGB + density

  // Number of (x, y, z) coordinates to compute
  uint32_t n_coords = image_width * image_height * n_samples;

  json config = {
      {"loss", {{"otype", "L2"}}},
      {"optimizer", {
                        {"otype", "Adam"},
                        {"learning_rate", 1e-5},
                        {"beta1", 0.9f},
                        {"beta2", 0.99f},
                    }},
      {"encoding", {
                       {"otype", "Identity"},
                   }},
      {"network", {
                      {"otype", "FullyFusedMLP"},
                      {"n_neurons", 64},
                      {"n_hidden_layers", 8},
                      {"activation", "ReLU"},
                      {"output_activation", "None"},
                  }},
  };

  json encoding_opts = config.value("encoding", json::object());
  json loss_opts = config.value("loss", json::object());
  json optimizer_opts = config.value("optimizer", json::object());
  json network_opts = config.value("network", json::object());

  std::shared_ptr<Loss<precision_t>> loss{create_loss<precision_t>(loss_opts)};
  std::shared_ptr<Optimizer<precision_t>> optimizer{create_optimizer<precision_t>(optimizer_opts)};
  std::shared_ptr<NetworkWithInputEncoding<precision_t>> network = std::make_shared<NetworkWithInputEncoding<precision_t>>(n_input_dims, n_output_dims, encoding_opts, network_opts);

  auto trainer = std::make_shared<Trainer<float, precision_t, precision_t>>(network, optimizer, loss);

  // 0. Load precomputed network weights
  vector<float> weights = load_weights("../nerfdata/tinylego/weights.txt");

  trainer->set_params_full_precision(weights.data(), weights.size());

  // 1. Load a known pose

  json transform_data = read_json("../nerfdata/tinylego/transforms.json");
  string dataset_path = "../nerfdata/tinylego";

  // In TinyNeRF dataset, camer_angle_x is the focal length (no need to calculate)
  float focal_length = transform_data["camera_angle_x"];
  auto [image_paths, c2ws] = get_image_c2w(transform_data, dataset_path);
  auto pose = c2ws[101];

  auto [ray_origins, ray_directions] = get_ray_bundle(image_width, image_height, focal_length, pose);
  auto [query_pts, depth_values] = compute_query_points_from_rays(ray_origins, ray_directions, near, far, n_samples);

  auto pts_flat = flatten_query_pts(query_pts);
  std::vector<float> host_pts_vec(pts_flat.begin(), pts_flat.end());
  GPUMemory<float> pts_vec(host_pts_vec.size());
  pts_vec.copy_from_host(host_pts_vec);
  std::cout << "Total number of (x, y, z) points for inference: " << host_pts_vec.size() / n_input_dims << std::endl;
  hipStream_t inference_stream;
  CUDA_CHECK_THROW(hipStreamCreate(&inference_stream));

  // Note: These are column-major
  GPUMatrix<float> inference_batch(pts_vec.data(), n_input_dims, n_coords);
  GPUMatrix<float> prediction(n_output_dims, n_coords);

  std::cout << "Beginning inference benchmark..." << std::endl;
  std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

  // Inference benchmark
  int n_iterations = 100;
  double mean_inference_throughput = 0;
  int mean_counter = 0;
  int print_interval = n_iterations / 10;
  int n_iterations_warmup = n_iterations / 2;

  for (uint32_t i = 0; i < n_iterations; ++i)
  {
    bool print_loss = i % print_interval == 0;

    // 2. Run inference
    network->inference(inference_stream, inference_batch, prediction);
    // std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    // auto microseconds = std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count();
    // std::cout << "Inference time=" << microseconds << "µs" << std::endl;
    render_output(prediction.data(),
                  n_coords,
                  image_height,
                  image_width,
                  n_samples,
                  n_output_dims,
                  depth_values);

    // Debug outputs
    if (print_loss)
    {
      hipDeviceSynchronize();
      std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
      auto microseconds = std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count();
      double throughput = print_interval * n_coords / ((double)microseconds / 1000000.0);
      std::cout << "Iteration#" << i << ": "
                << "time=" << microseconds << "[µs] thp=" << throughput << "/s" << std::endl;

      begin = end;

      if (i >= n_iterations_warmup)
      {
        mean_inference_throughput += throughput;
        ++mean_counter;
      }
    }
  }

  mean_inference_throughput /= (double)mean_counter;
  std::cout << "Finished inference benchmark. Mean throughput is " << mean_inference_throughput << "/s." << std::endl;

  return EXIT_SUCCESS;
}
