#include <tiny-cuda-nn/misc_kernels.h>
#include <tiny-cuda-nn/networks/fully_fused_mlp.h>

#include <tiny-cuda-nn/config.h>

#include <chrono>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <random>
#include <stdexcept>
#include <string>
#include <thread>
#include <vector>

#include "data.h"
#include "utils.h"
#include "nerf.h"
#include <xtensor/xio.hpp>
#include <xtensor/xmanipulation.hpp>
#include <xtensor/xoperation.hpp>
#include <algorithm>

using namespace tcnn;
using std::string;
using std::vector;
using precision_t = network_precision_t;

std::ostream &operator<<(std::ostream &o, tcnn::MatrixLayout c)
{
  std::cout << static_cast<int>(c);
  return o;
}

template <typename T>
void render_output(const T *output, int n_coords, int image_height, int image_width, int n_samples, int n_output_dims, xt::xtensor<float, 1UL, xt::layout_type::row_major> depth_values)
{
  std::vector<T> host_data(n_coords * n_output_dims);
  CUDA_CHECK_THROW(hipMemcpy(host_data.data(), output, host_data.size() * sizeof(T), hipMemcpyDeviceToHost));

  std::vector<float> float_host_data(host_data.size());
  for (size_t i = 0; i < host_data.size(); ++i)
  {
    float_host_data[i] = (float)host_data[i];
  }

  std::vector<std::size_t> rf_shape = {(size_t)image_height, (size_t)image_width, (size_t)n_samples, (size_t)n_output_dims};
  auto radiance_field = xt::adapt(float_host_data, rf_shape);
  auto rgb = render_rays(radiance_field, depth_values);
}

int main(int argc, char *argv[])
{

  // auto n_radiance_field = xt::load_npy<float>("../nerfdata/tinylego/n_rf.npy");
  // auto n_ray_origins = xt::load_npy<float>("../nerfdata/tinylego/n_rayso.npy");
  // auto n_depth_values = xt::load_npy<float>("../nerfdata/tinylego/n_dv.npy");
  // auto rgb = render_rays(n_radiance_field, n_ray_origins, n_depth_values);
  // return EXIT_SUCCESS;

  // 0. Load precomputed weights into network
  // 1. Load a known pose
  // 2. Run inference with the pose
  // 3. Perform volume rendering to output RGB image
  // 4. Write image to file

  const uint32_t image_width = 100;
  const uint32_t image_height = 100;
  const uint8_t n_frequencies = 6;
  const uint8_t n_samples = 64;
  const float near = 2.;
  const float far = 6.;
  // TinyNeRF only uses (xyz) -- no direction
  const uint32_t n_input_dims = 39;
  const uint32_t n_output_dims = 4; // RGB + density

  // Number of (x, y, z) coordinates to compute
  uint32_t n_coords = image_width * image_height * n_samples;

  // try
  // {
  json config = {
      {"loss", {{"otype", "L2"}}},
      {"optimizer", {
                        {"otype", "Adam"},
                        {"learning_rate", 1e-5},
                        {"beta1", 0.9f},
                        {"beta2", 0.99f},
                    }},
      {"encoding", {
                       {"otype", "Identity"},
                   }},
      {"network", {
                      {"otype", "FullyFusedMLP"},
                      {"n_neurons", 64},
                      {"n_hidden_layers", 8},
                      {"activation", "ReLU"},
                      {"output_activation", "None"},
                  }},
  };

  json encoding_opts = config.value("encoding", json::object());
  json loss_opts = config.value("loss", json::object());
  json optimizer_opts = config.value("optimizer", json::object());
  json network_opts = config.value("network", json::object());

  std::shared_ptr<Loss<precision_t>> loss{create_loss<precision_t>(loss_opts)};
  std::shared_ptr<Optimizer<precision_t>> optimizer{create_optimizer<precision_t>(optimizer_opts)};
  std::shared_ptr<NetworkWithInputEncoding<precision_t>> network = std::make_shared<NetworkWithInputEncoding<precision_t>>(n_input_dims, n_output_dims, encoding_opts, network_opts);

  auto trainer = std::make_shared<Trainer<float, precision_t, precision_t>>(network, optimizer, loss);

  // 0. Load precomputed network weights
  vector<float> weights = load_weights("../nerfdata/tinylego/weights.txt");
  // std::cout << "weights: " << weights.size() << std::endl;

  // for (auto &p : network->layer_sizes())
  // {
  //   std::cout << p.second << ", " << p.first << std::endl;
  // }
  trainer->set_params_full_precision(weights.data(), weights.size());

  // 1. Load a known pose

  json transform_data = read_json("../nerfdata/tinylego/transforms.json");
  string dataset_path = "../nerfdata/tinylego";
  // In TinyNeRF dataset, camer_angle_x is the focal length (no need to calculate)
  float focal_length = transform_data["camera_angle_x"];
  auto [image_paths, c2ws] = get_image_c2w(transform_data, dataset_path);
  auto pose = c2ws[20];

  auto [ray_origins, ray_directions] = get_ray_bundle(image_width, image_height, focal_length, pose);
  auto [query_pts, depth_values] = compute_query_points_from_rays(ray_origins, ray_directions, near, far, n_samples);

  auto pts_flat = flatten_query_pts(query_pts);
  std::vector<float> host_pts_vec(pts_flat.begin(), pts_flat.end());
  GPUMemory<float> pts_vec(host_pts_vec.size());
  pts_vec.copy_from_host(host_pts_vec);
  std::cout << "Total number of (x, y, z) points for inference: " << host_pts_vec.size() / n_input_dims << std::endl;

  hipStream_t inference_stream;
  CUDA_CHECK_THROW(hipStreamCreate(&inference_stream));

  // Note: These are column-major
  GPUMatrix<float> inference_batch(pts_vec.data(), n_input_dims, n_coords);
  GPUMatrix<float> prediction(n_output_dims, n_coords);

  // 2. Run inference
  network->inference(inference_stream, inference_batch, prediction);

  render_output(prediction.data(),
                n_coords,
                image_height,
                image_width,
                n_samples,
                n_output_dims,
                depth_values);

  return EXIT_SUCCESS;
}
